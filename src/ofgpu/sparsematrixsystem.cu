/***************************************************************************
    begin                : Fri Apr 1 2011
    copyright            : (C) 2011 Symscape
    website              : www.symscape.com
***************************************************************************/
/*
    This file is part of ofgpu.

    ofgpu is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    ofgpu is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with ofgpu.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <iostream>

#include "ofgpu/sparsematrixsystem.h"


namespace ofgpu
{
  SparseMatrixSystem::SparseMatrixSystem()
    : m_maxEntriesPerRow(0)
    , m_device(-1)
  {}
  
    
  SparseMatrixSystem::~SparseMatrixSystem()
  {}


  SparseMatrixSystem &
  SparseMatrixSystem::getSingleton()
  {
    // Skip cleaning up to avoid conflicts on Windows
    static SparseMatrixSystem* singleton = new SparseMatrixSystem;
    return *singleton;
  }


/*
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 *
 * Applies to _ConvertSMVer2Cores only
 */
// Beginning of GPU Architecture definitions
inline int _ConvertSMVer2Cores(int major, int minor)
{
    // Defines for GPU Architecture types (using the SM version to determine the # of cores per SM
    typedef struct
    {
        int SM; // 0xMm (hexidecimal notation), M = SM Major version, and m = SM minor version
        int Cores;
    } sSMtoCores;

    sSMtoCores nGpuArchCoresPerSM[] =
    {
        { 0x10,  8 }, // Tesla Generation (SM 1.0) G80 class
        { 0x11,  8 }, // Tesla Generation (SM 1.1) G8x class
        { 0x12,  8 }, // Tesla Generation (SM 1.2) G9x class
        { 0x13,  8 }, // Tesla Generation (SM 1.3) GT200 class
        { 0x20, 32 }, // Fermi Generation (SM 2.0) GF100 class
        { 0x21, 48 }, // Fermi Generation (SM 2.1) GF10x class
        { 0x30, 192}, // Kepler Generation (SM 3.0) GK10x class
        { 0x35, 192}, // Kepler Generation (SM 3.5) GK11x class
        {   -1, -1 }
    };

    int index = 0;

    while (nGpuArchCoresPerSM[index].SM != -1)
    {
        if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor))
        {
            return nGpuArchCoresPerSM[index].Cores;
        }

        index++;
    }

    // If we don't find the values, we default use the previous one to run properly
	std::cout << "MapSMtoCores for SM " << major << "." << minor << " is undefined. Default to use " << nGpuArchCoresPerSM[7].Cores << " Cores/SM" << std::endl;

    return nGpuArchCoresPerSM[7].Cores;
}
// end of GPU Architecture definitions


  void
  SparseMatrixSystem::setDevice(int const requestedDevice)
  {
    std::cout << "ofgpu CUDA info begin" << std::endl;

    if (0 <= m_device) {
      std::cout << "Warning: CUDA device already set as: " << m_device << std::endl;
      return;
    }

	int deviceCount = 0, driverVersion = 0, runtimeVersion = 0;
	hipGetDeviceCount(&deviceCount);

	std::cout << "Available CUDA devices" << std::endl;

	for (int i = 0; deviceCount > i; ++i) {
	  hipSetDevice(i);
	  hipDeviceProp_t prop;
	  hipGetDeviceProperties(&prop, i);
      hipDriverGetVersion(&driverVersion);
      hipRuntimeGetVersion(&runtimeVersion);

	  std::cout << "  Device: " << i << std::endl
	            << "    Name: \"" << prop.name << "\"" << std::endl
				<< "    Driver Version: " << driverVersion/1000 << "." << (driverVersion%100)/10 << std::endl
				<< "    Runtime Version: " << runtimeVersion/1000 << "." << (runtimeVersion%100)/10 << std::endl
	            << "    Capability: " << prop.major << "." << prop.minor << std::endl
	            << "    Processors: " << prop.multiProcessorCount << std::endl
	            << "    Cores: " << prop.multiProcessorCount * _ConvertSMVer2Cores(prop.major, prop.minor) << std::endl
	            << "    Memory (MBytes): " << (float)prop.totalGlobalMem/1048576.0f << std::endl
				<< "    GPU Clock rate (GHz): " << prop.clockRate * 1e-6f << std::endl;
	}

    m_device = requestedDevice;
    hipSetDevice(m_device);
    hipGetDevice(&m_device);

    if (requestedDevice != m_device) {
      std::cout << "Warning: Could not find requested CUDA device = " << requestedDevice << std::endl;
    }

	std::cout << "Selected CUDA device: " << m_device << std::endl
	          << "ofgpu CUDA info end" << std::endl
              << std::endl;
  }


  struct RowEntryFn
  {
    enum {LOW_FACE_START_i = 0, LOW_FACE_START_i_plus_1, UP_FACE_START_i, UP_FACE_START_i_plus_1, ROW_ENTRY_i};

    template <typename Tuple>
    __host__ __device__
    void operator() (Tuple t)
    {
        // rowEntry[i] = lowFacesStart[i + 1] - lowFacesStart[i] + 1 + upFacesStart[i + 1] - upFacesStart[i]
        thrust::get<ROW_ENTRY_i>(t) = thrust::get<LOW_FACE_START_i_plus_1>(t) - thrust::get<LOW_FACE_START_i>(t) + 1 + 
	                              thrust::get<UP_FACE_START_i_plus_1>(t)  - thrust::get<UP_FACE_START_i>(t);
    }
  };


  struct AssignIndicesFn
  {
    index const indicesRows;
    index const * const lowFaceToCell; 
    index const * const upFaceToCell;
    
    // Can't access overloaded operators in thrust, so need to get raw data for A
    index* const column_indices;

    index const X;
      
    AssignIndicesFn(Matrix & A,
		    IndexArray const & lowFaceToCell, 
		    IndexArray const & upFaceToCell)
      : indicesRows(A.column_indices.num_rows)
      , lowFaceToCell(lowFaceToCell.data().get())
      , upFaceToCell(upFaceToCell.data().get())
      , column_indices(A.column_indices.values.data().get())
      , X(Matrix::invalid_index)
    {}


    __host__ __device__
    index colIndex(index const i, index const j) const
    {
      // Mimics cusp::detail::index_of<index>(i, j, A.column_indices.num_rows, A.column_indices.num_cols, cusp::column_major())
      return i + j * indicesRows;
    }


    __host__ __device__
    void setLower(index const i, index const j, index const f)
    {
      column_indices[colIndex(i,j)] = lowFaceToCell[f];
    }


    __host__ __device__
    void setDiagonal(index const i, index const j)
    {
      column_indices[colIndex(i, j)] = i;
    }


    __host__ __device__
    void setUpper(index const i, index const j, index const f)
    {
      column_indices[colIndex(i,j)] = upFaceToCell[f];
    }


    __host__ __device__
    void setPadding(index const i, index const j)
    {
      column_indices[colIndex(i,j)] = X;
    }
  }; // struct AssignIndicesFn


  struct AssignValuesFn
  {
    index const valuesRows;
    real  const * const lowCellValueFromFace;
    real  const * const diagCellValue;
    real  const * const upCellValueFromFace;

    // Can't access overloaded operators in thrust, so need to get raw data for A
    real* const values;
 

    AssignValuesFn(Matrix & A,
		   RealArray const & lowCellValueFromFace,
		   RealArray const & diagCellValue,
		   RealArray const & upCellValueFromFace)
      : valuesRows(A.values.num_rows)
      , lowCellValueFromFace(lowCellValueFromFace.data().get())
      , diagCellValue(diagCellValue.data().get())
      , upCellValueFromFace(upCellValueFromFace.data().get())
      , values(A.values.values.data().get())
    {}


    __host__ __device__
    index valueIndex(index const i, index const j) const
    {
      //  Mimics cusp::detail::index_of<index>(i, j, A.values.num_rows, A.values.num_cols, cusp::column_major())
      return i + j * valuesRows;
    }


    __host__ __device__
    void setLower(index const i, index const j, index const f)
    {
      values[valueIndex(i, j)] = lowCellValueFromFace[f];
    }


    __host__ __device__
    void setDiagonal(index const i, index const j)
    {
      values[valueIndex(i, j)] = diagCellValue[i];
    }


    __host__ __device__
    void setUpper(index const i, index const j, index const f)
    {
       values[valueIndex(i, j)] = upCellValueFromFace[f];
    }


    __host__ __device__
    void setPadding(index const i, index const j)
    {
       values[valueIndex(i, j)] = 0.;
    }
  }; // struct AssignValuesFn


  template<class T_Assigner>
  struct MatrixAssignFn
  {
    T_Assigner assigner;
    index const maxEntriesPerRow;
    index const * const lowFace;
      
    enum {LOW_FACE_START_i = 0, LOW_FACE_START_i_plus_1, UP_FACE_START_i, UP_FACE_START_i_plus_1, ROW_i};


    MatrixAssignFn(T_Assigner const & assigner,
		   index const maxEntriesPerRow,
		   IndexArray const & lowFace)
      : assigner(assigner)
      , maxEntriesPerRow(maxEntriesPerRow)
      , lowFace(lowFace.data().get())
    {}


    template <typename Tuple>
    __host__ __device__
    void operator() (Tuple t)
    {
      index const rowI = thrust::get<ROW_i>(t);
      index entryJ     = 0;
      index face;

      index fBegin = thrust::get<LOW_FACE_START_i>(t);
      index fEnd   = thrust::get<LOW_FACE_START_i_plus_1>(t);

      for (index f = fBegin; f < fEnd; ++f) {
		face = lowFace[f];
		assigner.setLower(rowI, entryJ, face);
		++entryJ;
      }

      assigner.setDiagonal(rowI, entryJ);
      ++entryJ;

      fBegin = thrust::get<UP_FACE_START_i>(t);
      fEnd   = thrust::get<UP_FACE_START_i_plus_1>(t);

      for (index f = fBegin; f < fEnd; ++f) {
		assigner.setUpper(rowI, entryJ, f);
		++entryJ;
      }
     
      for (; entryJ < maxEntriesPerRow; ++entryJ) {
		assigner.setPadding(rowI, entryJ);
      }
    }
  };



  void     
  SparseMatrixSystem::initialize(SparseMatrixArgs const & args)
  {   
    m_lowFacesStart.assign(args.lowFacesStart, args.lowFacesStart + args.nCells + 1);
    m_upFacesStart.assign(args.upFacesStart, args.upFacesStart + args.nCells + 1);

    {
      IndexArray dRowEntries(args.nCells);
      
      // Count entries per row (cell)
      thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(m_lowFacesStart.begin(), ++m_lowFacesStart.begin(), 
								    m_upFacesStart.begin(), ++m_upFacesStart.begin(), 
								    dRowEntries.begin())),
		       thrust::make_zip_iterator(thrust::make_tuple(--m_lowFacesStart.end(), m_lowFacesStart.end(), 
								    --m_upFacesStart.end(), m_upFacesStart.end(), 
								    dRowEntries.end())),
		       RowEntryFn());
      
      // Total entry count
      index entryCount = thrust::reduce(dRowEntries.begin(), dRowEntries.end());
      
      // Max entries per row
      m_maxEntriesPerRow = *(thrust::max_element(dRowEntries.begin(), dRowEntries.end()));

      m_A.resize(args.nCells, args.nCells, entryCount, m_maxEntriesPerRow);
    }

    m_lowFace.assign(args.lowFace, args.lowFace + args.nFaces);

    IndexArray dLowFaceToCell(args.lowFaceToCell, args.lowFaceToCell + args.nFaces);
    IndexArray dUpFaceToCell(args.upFaceToCell, args.upFaceToCell + args.nFaces);
    thrust::counting_iterator<int> first(0);
    thrust::counting_iterator<int> last = first + args.nCells;
    
    thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(m_lowFacesStart.begin(), ++m_lowFacesStart.begin(), 
								  m_upFacesStart.begin() , ++m_upFacesStart.begin(),
								  first)),
                     thrust::make_zip_iterator(thrust::make_tuple(--m_lowFacesStart.end(), m_lowFacesStart.end(), 
								  --m_upFacesStart.end() , m_upFacesStart.end(),
								  last)),
                     MatrixAssignFn<AssignIndicesFn>(AssignIndicesFn(m_A, dLowFaceToCell, dUpFaceToCell),
						     m_maxEntriesPerRow,
						     m_lowFace));
  }


  void
  SparseMatrixSystem::assignMatrix(SparseMatrixArgs const & args)
  {
    RealArray dLowCellValueFromFace(args.lowCellValueFromFace, args.lowCellValueFromFace + args.nFaces);
    RealArray dDiagCellValue(args.diagCellValue, args.diagCellValue + args.nCells);
    RealArray dUpCellValueFromFace(args.upCellValueFromFace, args.upCellValueFromFace + args.nFaces);
    thrust::counting_iterator<int> first(0);
    thrust::counting_iterator<int> last = first + args.nCells;

    thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(m_lowFacesStart.begin(), ++m_lowFacesStart.begin(), 
								  m_upFacesStart.begin() , ++m_upFacesStart.begin(),
								  first)),
                     thrust::make_zip_iterator(thrust::make_tuple(--m_lowFacesStart.end(), m_lowFacesStart.end(), 
								  --m_upFacesStart.end() , m_upFacesStart.end(),
								  last)),
                     MatrixAssignFn<AssignValuesFn>(AssignValuesFn(m_A, dLowCellValueFromFace, dDiagCellValue, dUpCellValueFromFace),
						    m_maxEntriesPerRow,
						    m_lowFace));
  }


  void
  SparseMatrixSystem::update(SparseMatrixArgs const & args)
  {
    if (m_A.num_rows != args.nCells) {
      initialize(args);
    }
    
    assignMatrix(args);
  }
}
